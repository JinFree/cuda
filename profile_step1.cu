/* compile command

compile : nvcc -arch=sm_35  a.cu -o a.out
profile : nvprof -o log1.o ./a.out
view : nvvp log1.o
 
 */


#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void init_data_kernel( int n, double* x){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if ( i < n )
	{
		x[i] = n - i;
	}
}

__global__ void daxpy_kernel(int n, double a, double * x, double * y){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n)	{
		y[i] = a*x[i] + y[i];
	}
}

__global__ void check_results_kernel( int n, double correctvalue, double * x ){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n)	{
		if ( x[i] != correctvalue )
		{
			printf("ERROR at index = %d, expected = %f, actual: %f\n",i,correctvalue,x[i]);
		}
	}
}

void init_host_data( int n, double * x ){
	for (int i=0; i<n; ++i)	{
		x[i] = i;
	}
}

void init_data(int n, double* x, double* x_d, double* y_d){

	hipStream_t copy_stream;
	hipStream_t compute_stream;
	hipStreamCreate(&copy_stream);
	hipStreamCreate(&compute_stream);

	hipMemcpyAsync( x_d, x, n*sizeof(double), hipMemcpyDefault, copy_stream );
	init_data_kernel<<<ceil(n/256),256,0,compute_stream>>>(n, y_d);

	hipStreamSynchronize(copy_stream);
	hipStreamSynchronize(compute_stream);

	hipStreamDestroy(compute_stream);
	hipStreamDestroy(copy_stream);

}

void daxpy(int n, double a, double* x_d, double* y_d){
	daxpy_kernel<<<ceil(n/256),256>>>(n,a,x_d,y_d);
	hipDeviceSynchronize();
}

void check_results( int n, double correctvalue, double* x_d ){
	check_results_kernel<<<ceil(n/256),256>>>(n,correctvalue,x_d);
}

void run_test(int n){
	double* x;
	double* x_d;
	double* y_d;
	hipSetDevice(0);
	hipHostMalloc((void**) &x, n*sizeof(double), hipHostMallocDefault);
	hipMalloc((void**)&x_d,n*sizeof(double));
	hipMalloc((void**)&y_d,n*sizeof(double));

	init_host_data(n, x);

	init_data(n,x,x_d,y_d);

	daxpy(n,1.0,x_d,y_d);

	check_results(n, n, y_d);

	hipFree(y_d);
	hipFree(x_d);
	hipHostFree(x);
	hipDeviceSynchronize();

}

int main(){
	int n = 1<<22;
	run_test(n);
	return 0;
}
