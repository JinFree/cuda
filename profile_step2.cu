/* compile command
compile : nvcc -DUSE_NVTX -lnvToolsExt -arch=sm_35  a.cu -o a.out
profile : nvprof -o log1.o ./a.out
view : nvvp 
 */


#include <hip/hip_runtime.h>
#include <cstdio>

#ifdef USE_NVTX
#include "nvToolsExt.h"

const uint32_t colors[] = { 0x0000ff00, 0x000000ff, 0x00ffff00, 0x00ff00ff, 0x0000ffff, 0x00ff0000, 0x00ffffff };
const int num_colors = sizeof(colors)/sizeof(uint32_t);

#define PUSH_RANGE(name,cid) { \
	int color_id = cid; \
	color_id = color_id%num_colors;\
	nvtxEventAttributes_t eventAttrib = {0}; \
	eventAttrib.version = NVTX_VERSION; \
	eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE; \
	eventAttrib.colorType = NVTX_COLOR_ARGB; \
	eventAttrib.color = colors[color_id]; \
	eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
	eventAttrib.message.ascii = name; \
	nvtxRangePushEx(&eventAttrib); \
}
#define POP_RANGE nvtxRangePop();
#else
#define PUSH_RANGE(name,cid)
#define POP_RANGE
#endif

__global__ void init_data_kernel( int n, double* x){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if ( i < n )	{
		x[i] = n - i;
	}
}


__global__ void daxpy_kernel(int n, double a, double * x, double * y){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n)	{
		y[i] = a*x[i] + y[i];
	}
}

__global__ void check_results_kernel( int n, double correctvalue, double * x ){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n)
	{
		if ( x[i] != correctvalue )
		{
			printf("ERROR at index = %d, expected = %f, actual: %f\n",i,correctvalue,x[i]);
		}
	}
}

void init_host_data( int n, double * x ){
	PUSH_RANGE("init_host_data",1)
	for (int i=0; i<n; ++i)	{
		x[i] = i;
	}
	POP_RANGE
}

void init_data(int n, double* x, double* x_d, double* y_d){
	PUSH_RANGE("init_data",2)
	hipStream_t copy_stream;
	hipStream_t compute_stream;
	hipStreamCreate(&copy_stream);
	hipStreamCreate(&compute_stream);

	hipMemcpyAsync( x_d, x, n*sizeof(double), hipMemcpyDefault, copy_stream );
	init_data_kernel<<<ceil(n/256),256,0,compute_stream>>>(n, y_d);

	hipStreamSynchronize(copy_stream);
	hipStreamSynchronize(compute_stream);

	hipStreamDestroy(compute_stream);
	hipStreamDestroy(copy_stream);
	POP_RANGE
}

void daxpy(int n, double a, double* x_d, double* y_d){
	PUSH_RANGE("daxpy",3)
	daxpy_kernel<<<ceil(n/256),256>>>(n,a,x_d,y_d);
	hipDeviceSynchronize();
	POP_RANGE
}

void check_results( int n, double correctvalue, double* x_d ){
	PUSH_RANGE("check_results",4)
	check_results_kernel<<<ceil(n/256),256>>>(n,correctvalue,x_d);
	POP_RANGE
}

void run_test(int n){
	PUSH_RANGE("run_test",0)
	double* x;
	double* x_d;
	double* y_d;
	hipSetDevice(0);
	hipHostMalloc((void**) &x, n*sizeof(double), hipHostMallocDefault);
	hipMalloc((void**)&x_d,n*sizeof(double));
	hipMalloc((void**)&y_d,n*sizeof(double));

	init_host_data(n, x);

	init_data(n,x,x_d,y_d);

	daxpy(n,1.0,x_d,y_d);

	check_results(n, n, y_d);

	hipFree(y_d);
	hipFree(x_d);
	hipHostFree(x);
	hipDeviceSynchronize();
	POP_RANGE
}

int main(){
	int n = 1<<22;
	run_test(n);
	return 0;
}

